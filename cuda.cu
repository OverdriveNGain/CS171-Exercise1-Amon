#include <stdio.h>
#include <hip/hip_runtime.h>

__global__
void kernel_1t1e(float *matrixOut, float *matrix1, float *matrix2, int matrixDimLen){
    int length1D = matrixDimLen * matrixDimLen;
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < length1D) matrixOut[i]  = matrix1[i] + matrix2[i];
}

__global__
void kernel_1t1r(float *matrixOut, float *matrix1, float *matrix2, int matrixDimLen){
    int length1D = matrixDimLen * matrixDimLen;
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    for (int j = 0; j < matrixDimLen; j++)
        if (i + j < length1D) matrixOut[i + j]  = matrix1[i + j] + matrix2[i + j];
}

__global__
void kernel_1t1c(float *matrixOut, float *matrix1, float *matrix2, int matrixDimLen){
    int length1D = matrixDimLen * matrixDimLen;
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    for (int j = 0; j < matrixDimLen; j++){
        int index = i + j * matrixDimLen;
        if (index < length1D) matrixOut[index]  = matrix1[index] + matrix2[index];
    }
}

__host__
void matrixAdd(float*** output, float*** matrix1, float*** matrix2, int dimensionLength){
    int flattenedLength = dimensionLength*dimensionLength;
    size_t arrayByteSize = flattenedLength*sizeof(float);

    // Flatten input arrays
    float* matrix1Flat = (float*) malloc(arrayByteSize);
    float* matrix2Flat = (float*) malloc(arrayByteSize);
    float* outputFlat = (float*) malloc(arrayByteSize);

    // Copy data to flattened input arrays
    for (int i = 0; i < dimensionLength; i++){
        for (int j = 0; j < dimensionLength; j++){
            matrix1Flat[i * dimensionLength + j] = (*matrix1)[i][j];
            matrix2Flat[i * dimensionLength + j] = (*matrix2)[i][j];
            outputFlat[i * dimensionLength + j] = (*output)[i][j];
        }
    }

    // Allocate memory for arrays on device
    float *matrix1_d, *matrix2_d, *matrixOutput_d;
    hipMalloc(&matrix1_d, arrayByteSize);
    hipMalloc(&matrix2_d, arrayByteSize);
    hipMalloc(&matrixOutput_d, arrayByteSize);

    // Copy to device
    hipMemcpy(matrix1_d, matrix1Flat, arrayByteSize, hipMemcpyHostToDevice);
    hipMemcpy(matrix2_d, matrix2Flat, arrayByteSize, hipMemcpyHostToDevice);

    // Launch kernel
    // make more reasonable
    int threadBlockCount = ceil(flattenedLength/1024.0);
    int threadCountPerBlock = 1024;
    kernel_1t1e<<< threadBlockCount, threadCountPerBlock >>>(matrixOutput_d, matrix1_d, matrix2_d, dimensionLength);
    // kernel_1t1r<<< threadBlockCount, threadCountPerBlock >>>(matrixOutput_d, matrix1_d, matrix2_d, dimensionLength);
    // kernel_1t1c<<< threadBlockCount, threadCountPerBlock >>>(matrixOutput_d, matrix1_d, matrix2_d, dimensionLength);

    // Copy data from device output array to flattened host array
    hipMemcpy(outputFlat, matrixOutput_d, arrayByteSize, hipMemcpyDeviceToHost);

    // Copy data from flattened output array to output array
    for (int i = 0; i < dimensionLength; i++){
        for (int j = 0; j < dimensionLength; j++){
            (*output)[i][j] = outputFlat[i * dimensionLength + j];
        }
    }

    // Free GPU memory
    hipFree(matrix1_d);
    hipFree(matrix2_d);
    hipFree(matrixOutput_d);

    // Free flattened input arrays
    free(matrix1Flat);
    free(matrix2Flat);
    free(outputFlat);
}

__host__
void matrixAlloc(float** matrix, int dimensionLength){
    *matrix = (float*) malloc(dimensionLength*dimensionLength*sizeof(float));
}

__host__
void matrixPrint(float*** matrix, int dimensionLength){
    for (int i = 0; i < dimensionLength; i++){
        if (i == 0)
            printf("{{ ");
        else
            printf(" { ");
        
        for (int j = 0; j < dimensionLength; j++){
            if (j == dimensionLength - 1)
                printf("%f ", (*matrix)[i][j]);
            else
                printf("%f, ", (*matrix)[i][j]);
        }

        if (i == dimensionLength - 1)
            printf("}}\n");
        else
            printf("}\n");
    }
}

__host__
void matrixInitRandomValues(float*** matrix, int dimensionLength, float maxValue){
    for (int i = 0; i < dimensionLength; i++){
        for (int j = 0; j < dimensionLength; j++){
            // https://stackoverflow.com/questions/13408990/how-to-generate-random-float-number-in-c
            (*matrix)[i][j] = ((float)rand()/(float)(RAND_MAX)) * maxValue;
        }
    }
}

// Main program
int main()
{
    // Initialization of variables
    int dimensionLength = 3;
    size_t arrayByteSizeP = dimensionLength*sizeof(float*);
    size_t arrayByteSizeF = dimensionLength*sizeof(float);
    float **matrix1;
    float **matrix2;
    float **matrixOutput;

    // Memory allocation of matrices
    matrix1 = (float**) malloc(arrayByteSizeP);
    matrix2 = (float**) malloc(arrayByteSizeP);
    matrixOutput = (float**) malloc(arrayByteSizeP);
    for (int i = 0; i < dimensionLength; i++){
        matrix1[i] = (float*) malloc(arrayByteSizeF);
        matrix2[i] = (float*) malloc(arrayByteSizeF);
        matrixOutput[i] = (float*) malloc(arrayByteSizeF);
    }

    // Assignment of random values to matrix 1 and 2
    matrixInitRandomValues(&matrix1, dimensionLength, 100);
    matrixInitRandomValues(&matrix2, dimensionLength, 100);

    // Initial printing of values of matrix 1 and 2
    matrixPrint(&matrix1, dimensionLength);
    matrixPrint(&matrix2, dimensionLength);

    // Adding of matrices
    matrixAdd(&matrixOutput, &matrix1, &matrix2, dimensionLength);

    // Printing of output matrix
    matrixPrint(&matrixOutput, dimensionLength);

    // Free CPU memory
    for (int i = 0; i < dimensionLength; i++){
        free(matrix1[i]);
        free(matrix2[i]);
        free(matrixOutput[i]);
    }
    free(matrix1);
    free(matrix2);
    free(matrixOutput);

    printf("Success!\n");
    return 0;
}