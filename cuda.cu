#include <stdio.h>
#include <hip/hip_runtime.h>

__global__
void kernel_1t1e(float *matrixOut, float *matrix1, float *matrix2, int matrixDimLen){
    int length1D = matrixDimLen * matrixDimLen;
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < length1D) matrixOut[i]  = matrix1[i] + matrix2[i];
}

__global__
void kernel_1t1r(float *matrixOut, float *matrix1, float *matrix2, int matrixDimLen){
    int length1D = matrixDimLen * matrixDimLen;
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    for (int j = 0; j < matrixDimLen; j++)
        if (i + j < length1D) matrixOut[i + j]  = matrix1[i + j] + matrix2[i + j];
}

__global__
void kernel_1t1c(float *matrixOut, float *matrix1, float *matrix2, int matrixDimLen){
    int length1D = matrixDimLen * matrixDimLen;
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    for (int j = 0; j < matrixDimLen; j++){
        int index = i + j * matrixDimLen;
        if (index < length1D) matrixOut[index]  = matrix1[index] + matrix2[index];
    }
}

__host__
void matrixAdd(float*** output, float*** matrix1, float*** matrix2, int dimensionLength){
    int flattenedLength = dimensionLength*dimensionLength;
    size_t arrayByteSize = flattenedLength*sizeof(float);

    // Flatten input arrays
    float* matrix1Flat = (float*) malloc(arrayByteSize);
    float* matrix2Flat = (float*) malloc(arrayByteSize);
    float* outputFlat = (float*) malloc(arrayByteSize);

    // Copy data to flattened input arrays
    for (int i = 0; i < dimensionLength; i++){
        for (int j = 0; j < dimensionLength; j++){
            matrix1Flat[i * dimensionLength + j] = (*matrix1)[i][j];
            matrix2Flat[i * dimensionLength + j] = (*matrix2)[i][j];
            outputFlat[i * dimensionLength + j] = (*output)[i][j];
        }
    }

    // Allocate memory for arrays on device
    float *matrix1_d, *matrix2_d, *matrixOutput_d;
    hipMalloc(&matrix1_d, arrayByteSize);
    hipMalloc(&matrix2_d, arrayByteSize);
    hipMalloc(&matrixOutput_d, arrayByteSize);

    // Copy to device
    hipMemcpy(matrix1_d, matrix1Flat, arrayByteSize, hipMemcpyHostToDevice);
    hipMemcpy(matrix2_d, matrix2Flat, arrayByteSize, hipMemcpyHostToDevice);

    // Launch kernel
    // make more reasonable
    int threadBlockCount = ceil(flattenedLength/1024.0);
    int threadCountPerBlock = 1024;
    kernel_1t1e<<< threadBlockCount, threadCountPerBlock >>>(matrixOutput_d, matrix1_d, matrix2_d, dimensionLength);
    // kernel_1t1r<<< threadBlockCount, threadCountPerBlock >>>(matrixOutput_d, matrix1_d, matrix2_d, dimensionLength);
    // kernel_1t1c<<< threadBlockCount, threadCountPerBlock >>>(matrixOutput_d, matrix1_d, matrix2_d, dimensionLength);

    // Copy data from device output array to flattened host array
    hipMemcpy(outputFlat, matrixOutput_d, arrayByteSize, hipMemcpyDeviceToHost);

    // Copy data from flattened output array to output array
    for (int i = 0; i < dimensionLength; i++){
        for (int j = 0; j < dimensionLength; j++){
            (*output)[i][j] = outputFlat[i * dimensionLength + j];
        }
    }

    // Free GPU memory
    hipFree(matrix1_d);
    hipFree(matrix2_d);
    hipFree(matrixOutput_d);

    // Free flattened input arrays
    free(matrix1Flat);
    free(matrix2Flat);
    free(outputFlat);
}

__host__
void matrixAlloc(float** matrix, int dimensionLength){
    *matrix = (float*) malloc(dimensionLength*dimensionLength*sizeof(float));
}

__host__
void matrixPrint(float*** matrix, int dimensionLength){
    for (int i = 0; i < dimensionLength; i++){
        if (i == 0)
            printf("{{ ");
        else
            printf(" { ");
        
        for (int j = 0; j < dimensionLength; j++){
            if (j == dimensionLength - 1)
                printf("%f ", (*matrix)[i][j]);
            else
                printf("%f, ", (*matrix)[i][j]);
        }

        if (i == dimensionLength - 1)
            printf("}}\n");
        else
            printf("}\n");
    }
}

__host__
void matrixInitRandomValues(float*** matrix, int dimensionLength, float maxValue){
    for (int i = 0; i < dimensionLength; i++){
        for (int j = 0; j < dimensionLength; j++){
            // https://stackoverflow.com/questions/13408990/how-to-generate-random-float-number-in-c
            (*matrix)[i][j] = ((float)rand()/(float)(RAND_MAX)) * maxValue;
        }
    }
}

// Main program
int main()
{
    int dimensionLength = 3;
    size_t arrayByteSizeP = dimensionLength*sizeof(float*);
    size_t arrayByteSizeF = dimensionLength*sizeof(float);
    float **matrix1;
    float **matrix2;
    float **matrixOutput;

    matrix1 = (float**) malloc(arrayByteSizeP);
    matrix2 = (float**) malloc(arrayByteSizeP);
    matrixOutput = (float**) malloc(arrayByteSizeP);
    for (int i = 0; i < dimensionLength; i++){
        matrix1[i] = (float*) malloc(arrayByteSizeF);
        matrix2[i] = (float*) malloc(arrayByteSizeF);
        matrixOutput[i] = (float*) malloc(arrayByteSizeF);
    }

    matrixInitRandomValues(&matrix1, dimensionLength, 100);
    matrixInitRandomValues(&matrix2, dimensionLength, 100);

    matrixPrint(&matrix1, dimensionLength);
    matrixPrint(&matrix2, dimensionLength);

    matrixAdd(&matrixOutput, &matrix1, &matrix2, dimensionLength);

    matrixPrint(&matrixOutput, dimensionLength);

    // Free CPU memory
    for (int i = 0; i < dimensionLength; i++){
        free(matrix1[i]);
        free(matrix2[i]);
        free(matrixOutput[i]);
    }
    free(matrix1);
    free(matrix2);
    free(matrixOutput);

    printf("SUCCESS!\n");
    return 0;
}